#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"

#define FILTER_RADIUS 2
#define TILE_DIM 32

//F: Convolution filter array pointer (2D- not linearized)
__constant__ float F[2*FILTER_RADIUS+1][2*FILTER_RADIUS+1];

__global__ void 2D_convolution_kernel(float *N, float *P,int width, int height){
  //N: Input array pointer (linearlized)
  //P: Output array pointer (linearized)
  //width: width of the input/output array
  // height: height of the input/output array

  int row = blockIdx.y * TILE_DIM + threadIdx.y;
  int col = blockIdx.x * TILE_DIM + threadIdx.x;

  __shared__ N_s[TILE_DIM][TILE_DIM];
  if(row<height && col <width){
    N_s[threadIdx.y][threadIdx.x] = N[row*width+col];
  }
  else{
    N_s[threadIdx.y][threadIdx.x] = 0.0f;
  }
  __synchthreads();
  if(row<height && col<width){
    float Pval = 0.0f;
    for(int rowOffset = 0; rowOffset< 2*FILTER_RADIUS+1; rowOffset++){
      for(int colOffset = 0; colOffset< 2*FILTER_RADIUS+1; colOffset++){
        if(threadIdx.y-FILTER_RADIUS+rowOffset>=0 && threadIdx.y-FILTER_RADIUS+rowOffset < TILE_DIM && threadIdx.x-FILTER_RADIUS+colOffset >=0 && threadIdx.x-FILTER_RADIUS+colOffset < TILE_DIM){
          Pval+= F[rowOffset][colOffset]*N_s[threadIdx.y-FILTER_RADIUS+rowOffset][threadIdx.x-FILTER_RADIUS+colOffset];
        }
        else{
          currRow = row - FILTER_RADIUS + rowOffset;
          currCol = col - FILTER_RADIUS + colOffset;
          if (currRow >= 0 && currRow<height && currCol>=0 && currCol <width){
            Pval+= N[currRow*width+currCol]*F[rowOffset][colOffset];
          }
        }
      }
    }
    P[row*width+col] = Pval;
  }
} 


int main(){
  float* F_h = (float*)malloc(img_size);
  //Informs CUDA runtime that the data being copied into constand mem will not be changed during execution
  hipMemcpyToSymbol(HIP_SYMBOL(F),F_h,(2*FILTER_RADIUS+1)*(2*FILTER_RADIUS+1)*(sizeof(float)); 
  dim3 dimBlock(16,16,1);
  dim3 dimGrid(ceil(n/16.0), ceil(n/16.0),1);
  2D_convolution_kernel<<dimGrid,dimBlock>>
}
