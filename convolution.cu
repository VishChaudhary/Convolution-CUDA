#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"


__global__ void 2D_convolution_kernel(float *N, float *F, float *P, int r, int width, int height){
  //N: Input array pointer (linearlized)
  //F: Convolution filter array pointer (2D- not linearized)
  //P: Output array pointer (linearized)
  //r: Filter radius (2r+1)
  //width: width of the input/output array
  // height: height of the input/output array
  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float Pval = 0.0f;
  for(int rowOffset = 0; rowOffset< 2r+1; rowOffset++){
    for(int colOffset = 0; colOffset< 2r+1; colOffset++){
      currRow = row - r + rowOffset;
      currCol = col - r + colOffset;
      (if currRow >= 0 && currRow<height && currCol>=0 && currCol <width){
        Pval+= N[currRow*width+currCol]*F[rowOffset][colOffset];
      }
    }
  }
  P[row*width+col] = Pval;
} 


int main(){

  dim3 dimBlock(16,16,1);
  dim3 dimGrid(ceil(n/16.0), ceil(n/16.0),1);
  2D_convolution_kernel<<dimGrid,dimBlock>>
}
